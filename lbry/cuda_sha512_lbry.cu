#include "hip/hip_runtime.h"
/**
 * sha-512 CUDA implementation.
 * Tanguy Pruvot and Provos Alexis - JUL 2016
 */

//#define USE_ROT_ASM_OPT 0
#include <cuda_helper.h>
#include "cuda_vectors.h"
#include "miner.h"

static __constant__ uint64_t K_512[80] = {
	0x428A2F98D728AE22, 0x7137449123EF65CD, 0xB5C0FBCFEC4D3B2F, 0xE9B5DBA58189DBBC,	0x3956C25BF348B538, 0x59F111F1B605D019, 0x923F82A4AF194F9B, 0xAB1C5ED5DA6D8118,
	0xD807AA98A3030242, 0x12835B0145706FBE, 0x243185BE4EE4B28C, 0x550C7DC3D5FFB4E2,	0x72BE5D74F27B896F, 0x80DEB1FE3B1696B1, 0x9BDC06A725C71235, 0xC19BF174CF692694,
	0xE49B69C19EF14AD2, 0xEFBE4786384F25E3, 0x0FC19DC68B8CD5B5, 0x240CA1CC77AC9C65,	0x2DE92C6F592B0275, 0x4A7484AA6EA6E483, 0x5CB0A9DCBD41FBD4, 0x76F988DA831153B5,
	0x983E5152EE66DFAB, 0xA831C66D2DB43210, 0xB00327C898FB213F, 0xBF597FC7BEEF0EE4,	0xC6E00BF33DA88FC2, 0xD5A79147930AA725, 0x06CA6351E003826F, 0x142929670A0E6E70,
	0x27B70A8546D22FFC, 0x2E1B21385C26C926, 0x4D2C6DFC5AC42AED, 0x53380D139D95B3DF,	0x650A73548BAF63DE, 0x766A0ABB3C77B2A8, 0x81C2C92E47EDAEE6, 0x92722C851482353B,
	0xA2BFE8A14CF10364, 0xA81A664BBC423001, 0xC24B8B70D0F89791, 0xC76C51A30654BE30,	0xD192E819D6EF5218, 0xD69906245565A910, 0xF40E35855771202A, 0x106AA07032BBD1B8,
	0x19A4C116B8D2D0C8, 0x1E376C085141AB53, 0x2748774CDF8EEB99, 0x34B0BCB5E19B48A8,	0x391C0CB3C5C95A63, 0x4ED8AA4AE3418ACB, 0x5B9CCA4F7763E373, 0x682E6FF3D6B2B8A3,
	0x748F82EE5DEFB2FC, 0x78A5636F43172F60, 0x84C87814A1F0AB72, 0x8CC702081A6439EC,	0x90BEFFFA23631E28, 0xA4506CEBDE82BDE9, 0xBEF9A3F7B2C67915, 0xC67178F2E372532B,
	0xCA273ECEEA26619C, 0xD186B8C721C0C207, 0xEADA7DD6CDE0EB1E, 0xF57D4F7FEE6ED178,	0x06F067AA72176FBA, 0x0A637DC5A2C898A6, 0x113F9804BEF90DAE, 0x1B710B35131C471B,
	0x28DB77F523047D84, 0x32CAAB7B40C72493, 0x3C9EBE0A15C9BEBC, 0x431D67C49C100D4C,	0x4CC5D4BECB3E42B6, 0x597F299CFC657E2A, 0x5FCB6FAB3AD6FAEC, 0x6C44198C4A475817
};

#undef xor3
#define xor3(a,b,c) (a^b^c)

#define bsg5_0(x) xor3(ROTR64(x,28),ROTR64(x,34),ROTR64(x,39))
#define bsg5_1(x) xor3(ROTR64(x,14),ROTR64(x,18),ROTR64(x,41))
#define ssg5_0(x) xor3(ROTR64(x,1),ROTR64(x,8),shr_u64(x,7))
#define ssg5_1(x) xor3(ROTR64(x,19),ROTR64(x,61),shr_u64(x,6))


#define andor64(a,b,c) ((a & (b | c)) | (b & c))
#define xandx64(e,f,g) (g ^ (e & (g ^ f)))

__device__ __forceinline__
static void sha512_step2(uint64_t *const r,const uint64_t W,const uint64_t K, const int ord){

	const uint64_t T1 = r[(15-ord) & 7] + K + W + bsg5_1(r[(12-ord) & 7]) + xandx64(r[(12-ord) & 7],r[(13-ord) & 7],r[(14-ord) & 7]);
	r[(15-ord)& 7] = T1 + andor64(r[( 8-ord) & 7],r[( 9-ord) & 7],r[(10-ord) & 7]) + bsg5_0(r[( 8-ord) & 7]);
	r[(11-ord)& 7]+= T1;
}

/**************************************************************************************************/

__global__ __launch_bounds__(512,2)
void lbry_sha512_gpu_hash_32(const uint32_t threads, uint32_t *const __restrict__ g_hash)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	const uint64_t IV512[8] = {
		0x6A09E667F3BCC908, 0xBB67AE8584CAA73B, 0x3C6EF372FE94F82B, 0xA54FF53A5F1D36F1,
		0x510E527FADE682D1, 0x9B05688C2B3E6C1F, 0x1F83D9ABFB41BD6B, 0x5BE0CD19137E2179
	};
	uint64_t r[8];
	uint64_t W[16];
	if (thread < threads)
	{
		uint32_t *const pHash = &g_hash[thread<<4];

		*(uint2x4*)&r[ 0] = *(uint2x4*)&IV512[ 0];
		*(uint2x4*)&r[ 4] = *(uint2x4*)&IV512[ 4];

		*(uint2x4*)&W[ 0] = __ldg4((uint2x4*)&pHash[ 0]);
		
		W[4] = 0x8000000000000000; // end tag

		#pragma unroll
		for (uint32_t i = 5; i < 15; i++) W[i] = 0;

		W[15] = 0x100; // 256 bits

		uint64_t t1;
		uint64_t constants[2];		
		#pragma unroll 8
		for (int i = 0; i < 16; i+=2){
			*(uint4*)&constants = *(uint4*)&K_512[i];

			t1 = W[i+0] + r[ 7] + bsg5_1(r[ 4]) + xandx64(r[ 4], r[ 5], r[ 6]) + constants[0];
			#pragma unroll
			for (int l = 6; l >= 0; l--) r[l + 1] = r[l];
			r[0] = t1 + andor64(r[ 1], r[ 2], r[ 3]) + bsg5_0(r[ 1]);
			r[4]+= t1;
			
			t1 = W[i+1] + r[ 7] + bsg5_1(r[ 4]) + xandx64(r[ 4], r[ 5], r[ 6]) + constants[1];
			#pragma unroll
			for (int l = 6; l >= 0; l--) r[l + 1] = r[l];
			r[0] = t1 + andor64(r[ 1], r[ 2], r[ 3]) + bsg5_0(r[ 1]);
			r[4]+= t1;
		}

		#pragma unroll
		for (uint32_t i = 16; i < 80; i+=16){
			#pragma unroll 16
			for (uint32_t j = 0; j<16; j++){
				W[j & 15] += ssg5_0(W[(j - 15) & 15]) + W[(j - 7) & 15] + ssg5_1(W[(j - 2) & 15]);
			}
			#pragma unroll 8
			for (uint32_t j = 0; j<16; j+=2){
				*(uint4*)&constants = *(uint4*)&K_512[i+j];
				
				sha512_step2(r, W[j+0],constants[0], (i+j+0)&7);
				sha512_step2(r, W[j+1],constants[1], (i+j+1)&7);
			}
		}

		#pragma unroll 8
		for (uint32_t i = 0; i < 8; i++)
			r[i] = cuda_swab64(r[i] + IV512[i]);
		
		*(uint2x4*)&pHash[ 0] = *(uint2x4*)&r[ 0];
		*(uint2x4*)&pHash[ 8] = *(uint2x4*)&r[ 4];

	}
}

__host__
void lbry_sha512_hash_32(int thr_id, uint32_t threads, uint32_t *d_hash)
{
	const int threadsperblock = 512;

	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	lbry_sha512_gpu_hash_32 <<<grid, block>>> (threads, d_hash);
}


#define sph_u64 uint64_t


static const __constant__ __align__(16) ulong K512[80] =
{
	0x428A2F98D728AE22UL, 0x7137449123EF65CDUL,
	0xB5C0FBCFEC4D3B2FUL, 0xE9B5DBA58189DBBCUL,
	0x3956C25BF348B538UL, 0x59F111F1B605D019UL,
	0x923F82A4AF194F9BUL, 0xAB1C5ED5DA6D8118UL,
	0xD807AA98A3030242UL, 0x12835B0145706FBEUL,
	0x243185BE4EE4B28CUL, 0x550C7DC3D5FFB4E2UL,
	0x72BE5D74F27B896FUL, 0x80DEB1FE3B1696B1UL,
	0x9BDC06A725C71235UL, 0xC19BF174CF692694UL,
	0xE49B69C19EF14AD2UL, 0xEFBE4786384F25E3UL,
	0x0FC19DC68B8CD5B5UL, 0x240CA1CC77AC9C65UL,
	0x2DE92C6F592B0275UL, 0x4A7484AA6EA6E483UL,
	0x5CB0A9DCBD41FBD4UL, 0x76F988DA831153B5UL,
	0x983E5152EE66DFABUL, 0xA831C66D2DB43210UL,
	0xB00327C898FB213FUL, 0xBF597FC7BEEF0EE4UL,
	0xC6E00BF33DA88FC2UL, 0xD5A79147930AA725UL,
	0x06CA6351E003826FUL, 0x142929670A0E6E70UL,
	0x27B70A8546D22FFCUL, 0x2E1B21385C26C926UL,
	0x4D2C6DFC5AC42AEDUL, 0x53380D139D95B3DFUL,
	0x650A73548BAF63DEUL, 0x766A0ABB3C77B2A8UL,
	0x81C2C92E47EDAEE6UL, 0x92722C851482353BUL,
	0xA2BFE8A14CF10364UL, 0xA81A664BBC423001UL,
	0xC24B8B70D0F89791UL, 0xC76C51A30654BE30UL,
	0xD192E819D6EF5218UL, 0xD69906245565A910UL,
	0xF40E35855771202AUL, 0x106AA07032BBD1B8UL,
	0x19A4C116B8D2D0C8UL, 0x1E376C085141AB53UL,
	0x2748774CDF8EEB99UL, 0x34B0BCB5E19B48A8UL,
	0x391C0CB3C5C95A63UL, 0x4ED8AA4AE3418ACBUL,
	0x5B9CCA4F7763E373UL, 0x682E6FF3D6B2B8A3UL,
	0x748F82EE5DEFB2FCUL, 0x78A5636F43172F60UL,
	0x84C87814A1F0AB72UL, 0x8CC702081A6439ECUL,
	0x90BEFFFA23631E28UL, 0xA4506CEBDE82BDE9UL,
	0xBEF9A3F7B2C67915UL, 0xC67178F2E372532BUL,
	0xCA273ECEEA26619CUL, 0xD186B8C721C0C207UL,
	0xEADA7DD6CDE0EB1EUL, 0xF57D4F7FEE6ED178UL,
	0x06F067AA72176FBAUL, 0x0A637DC5A2C898A6UL,
	0x113F9804BEF90DAEUL, 0x1B710B35131C471BUL,
	0x28DB77F523047D84UL, 0x32CAAB7B40C72493UL,
	0x3C9EBE0A15C9BEBCUL, 0x431D67C49C100D4CUL,
	0x4CC5D4BECB3E42B6UL, 0x597F299CFC657E2AUL,
	0x5FCB6FAB3AD6FAECUL, 0x6C44198C4A475817UL
};

#define BSG5_1 bsg5_1
#define BSG5_0 bsg5_0
#define SSG5_1 ssg5_1
#define SSG5_0 ssg5_0

#define MAJ andor64
#define CH  xandx64

__device__ __forceinline__ void SHA2_512_STEP2(const ulong *W, uint ord, ulong *r, int i, const ulong *k)
{
	ulong T1;
	int x = 8 - ord;
	
	ulong a = r[x & 7], b = r[(x + 1) & 7], c = r[(x + 2) & 7], d = r[(x + 3) & 7];
	ulong e = r[(x + 4) & 7], f = r[(x + 5) & 7], g = r[(x + 6) & 7], h = r[(x + 7) & 7];
	
	T1 = h + BSG5_1(e) + CH(e, f, g) + W[i] + (K512[i]);
//	T1 = h + BSG5_1(e) + CH(e, f, g) + W[i] + (k[ord]);
	r[(3 + x) & 7] = d + T1;
	r[(7 + x) & 7] = T1 + BSG5_0(a) + MAJ(a, b, c);
}

__device__ __forceinline__ void SHA512Block(ulong *W, ulong *buf, const ulong *k)
{
//	ulong W[80]
	ulong  r[8];

	ulong constants[8];

	
//#pragma unroll 8
	for(int i = 0; i < 8; ++i) r[i] = buf[i];
	
	
	#pragma unroll 16
	for(int i = 16; i < 80; ++i) W[i] = SSG5_1(W[i - 2]) + W[i - 7] + SSG5_0(W[i - 15]) + W[i - 16];
	
//	#pragma unroll 2
	for(int i = 0; i < 80; i += 8)
	{
//		*(uint2x4*)&constants[0] = *(uint2x4*)&k[i];
//		*(uint2x4*)&constants[4] = *(uint2x4*)&k[i+4];
		#pragma unroll 8
		for(int j = 0; j < 8; ++j)
		{
			SHA2_512_STEP2(W, j, r, i + j, k);
//			SHA2_512_STEP2(W, j, r, i + j, &constants[0]);
		}
	}
//	#pragma unroll 8 
	for(int i = 0; i < 8; ++i) buf[i] += r[i];
}

#define TPB_SHA 176
#define TH_SH 76
__global__ __launch_bounds__(TPB_SHA,1)
__global__ void sha512_gpu_hash_64(int threads,  uint64_t *g_hash)
{
    int thread = (blockDim.x * blockIdx.x + threadIdx.x);
    if (thread < threads)
    {
        uint64_t *inpHash = &g_hash[thread<<3];

		// sha512
__shared__ sph_u64 Ws[TH_SH*80];
sph_u64 Wm[80];
 sph_u64 *W;
 // sph_u64 W[80];
if(threadIdx.x < TH_SH)
 W=&Ws[80*threadIdx.x];
else
 W=&Wm[0];


  sph_u64 state[8];
/*
		uint2x4 *phash = (uint2x4*)inpHash;
		uint2x4 *outpt = (uint2x4*)W;
		outpt[0] = __ldg4(&phash[0]);
		outpt[1] = __ldg4(&phash[1]);

#pragma unroll 8
  for (int i = 0; i < 8; i++)  W[i] = cuda_swab64(W[i]);
*/


#pragma unroll 8
  for (int i = 0; i < 8; i++)
    W[i] = cuda_swab64(__ldg(&inpHash[i]));


#pragma unroll 8
  for (int i = 8; i < 16; i++)
    W[i] = 0;

  state[0] = SPH_C64(0x6A09E667F3BCC908);
  state[1] = SPH_C64(0xBB67AE8584CAA73B);
  state[2] = SPH_C64(0x3C6EF372FE94F82B);
  state[3] = SPH_C64(0xA54FF53A5F1D36F1);
  state[4] = SPH_C64(0x510E527FADE682D1);
  state[5] = SPH_C64(0x9B05688C2B3E6C1F);
  state[6] = SPH_C64(0x1F83D9ABFB41BD6B);
  state[7] = SPH_C64(0x5BE0CD19137E2179);

  SHA512Block(W, state,K512);


  W[0] = 0x8000000000000000UL;
  W[1] = 0x0000000000000000UL;
  W[2] = 0x0000000000000000UL;
  W[3] = 0x0000000000000000UL;
  W[4] = 0x0000000000000000UL;
  W[5] = 0x0000000000000000UL;
  W[6] = 0x0000000000000000UL;
  W[7] = 0x0000000000000000UL;
  W[8] = 0x0000000000000000UL;
  W[9] = 0x0000000000000000UL;
  W[10] = 0x0000000000000000UL;
  W[11] = 0x0000000000000000UL;
  W[12] = 0x0000000000000000UL;
  W[13] = 0x0000000000000000UL;
  W[14] = 0x0000000000000000UL;
  W[15] = 0x0000000000000400UL;

  SHA512Block(W, state,K512);
#pragma unroll 8
  for (int i = 0; i < 8; i++)
    inpHash[i] = cuda_swab64(state[i]);
	}
}



__host__ void xevan_sha512_cpu_hash_64(int thr_id, int threads, uint32_t *d_hash)
{

	const int threadsperblock = TPB_SHA;

	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);
	sha512_gpu_hash_64<<<grid, block>>>(threads,  (uint64_t*)d_hash);

}
